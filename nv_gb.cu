#include "hip/hip_runtime.h"
#include "bootrom.h"
#include "gb_defs.h"
#include <stdio.h>
#include <sys/time.h>
#define CHECK_ERR_CUDA(err) if (err != hipSuccess) { printf("%s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }
//#define DEBUG

double get_time() {
  struct timeval tv; gettimeofday(&tv, NULL);
  return (tv.tv_sec + tv.tv_usec * 1e-6);
}

int read_cart(u8* cart, const char* fname) {
  FILE * file = fopen(fname, "r+");
  if (file == NULL || cart == NULL) return - 1;
  fseek(file, 0, SEEK_END);
  long int size = ftell(file);
  fclose(file);
  file = fopen(fname, "r+");
  int bytes_read = fread(cart, sizeof(u8), size, file);
  printf("read file %s, %d bytes\n", fname, bytes_read);
  return 0; fclose(file);
}
// CPU cycles LUT
__device__ u8 mcycles[256] = {
   4, 12,  8,  8,  4,  4,  8,  4, 20,  8,  8,  8,  4,  4,  8,  4,  // 00-0f
   4, 12,  8,  8,  4,  4,  8,  4, 12,  8,  8,  8,  4,  4,  8,  4,  // 10-1f
  12, 12,  8,  8,  4,  4,  8,  4, 12,  8,  8,  8,  4,  4,  8,  4,  // 20-2f
  12, 12,  8,  8, 12, 12, 12,  4, 12,  8,  8,  8,  4,  4,  8,  4,  // 30-3f
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // 40-4f
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // 50-5f
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // 60-6f
   8,  8,  8,  8,  8,  8,  4,  8,  4,  4,  4,  4,  4,  4,  8,  4,  // 70-7f
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // 80-8f
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // 90-9f
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // a0-af
   4,  4,  4,  4,  4,  4,  8,  4,  4,  4,  4,  4,  4,  4,  8,  4,  // b0-bf
  20, 12, 16, 16, 24, 16,  8, 16, 20, 16, 16,  4, 24, 24,  8, 16,  // c0-cf
  20, 12, 16,  0, 24, 16,  8, 16, 20, 16, 16,  0, 24,  0,  8, 16,  // d0-df
  12, 12,  8,  0,  0, 16,  8, 16, 16,  4, 16,  0,  0,  0,  8, 16,  // e0-ef
  12, 12,  8,  4,  0, 16,  8, 16, 12,  8, 16,  4,  0,  0,  8, 16}; // f0-ff

// 8-bit write
__device__ void w8(gb* g, u16 a, u8 v) {
  switch (a & 0xf000) {
    case 0x8000: // video ram 0x0000-0x0fff
    case 0x9000: // video ram 0x1000-0x1fff
#ifdef DEBUG
      //printf("__w8 VRAM %04x %04x %02x\n", a, a & 0x1fff, v);
#endif
      g->vram[a & 0x1fff] = v;
#ifdef DEBUG
      //printf("%04x %04x %02x\n", a, a & 0x1fff, v);
#endif
      break;
    case 0xc000:
    case 0xd000:
    case 0xe000:
      g->ram[a & 0x1fff] = v; break;
    case 0xf000:
#ifdef DEBUG
      //printf("__w8 HRAM %04x %04x %02x\n", a, a & 0x00ff, v);
#endif
      if (a < 0xfe00) g->ram[a & 0x1fff] = v;
      //if (a < 0xff00) g->oam[a & 0xff] = v;
      //else {
      if (a >= 0xff00) {
        //printf("%04x <- %02x\n", a, v);
        g->hram[a & 0xff] = v;
      }
      //  if (a == 0xff46) { oam_ram(g); }
      break;
    default:
#ifdef DEBUG
      //printf("w8 unhandled %04x\n", a & 0xf000);
#endif
      break;
  }
}
// 16-bit write
__device__ void w16(gb* g, u16 a, u16 v) { w8(g,a,v&0xff); w8(g,a+1,v>>8); }
// 8-bit read
__device__ u8 r8(gb* g, u16 a) {
  if (a <= 0x3fff) return g->rom[a & 0x3fff]; // rom bank 0
  else if (a <= 0x7fff) return g->rom[a & 0x7fff]; // rom bank 1
  else if (a >= 0xc000 && a <= 0xfdff) return g->ram[a & 0x1fff]; // internal ram
  else if (a == 0xff44) return 0x90;
  else if (a >= 0x8000 && a <= 0x9fff) { /*printf("0x%04x = %4x\n", a, g->vram[a & 0x1fff]);*/ return g->vram[a & 0x1fff]; }
  else if (a >= 0xff00 && a <= 0xffff) { return g->hram[a & 0xff]; }
  else {
#ifdef DEBUG
    //printf("r8 a = 0x%04x unhandled\n", a);
#endif
    return 0;
  }
}
// 16-bit read
__device__ u16 r16(gb* g, u16 a) { return ((u16)(r8(g, a+1)) << 8) | (u16)(r8(g, a)); } // read 2 bytes
// operand fetch
__device__ u8 f8(gb *g) { u8 r = r8(g, PC); PC+=1; return r;  } // fetch operand data (byte)
// 16-bit
__device__ u16 f16(gb* g) { u16 r = r16(g, PC); PC+=2; return r; } // fetch operand data (2 bytes)
// 16-bit stack push/pop
__device__ void push16(gb* g, u16 v) { SP -= 2; w16(g, SP, v); } // push onto the stack
__device__ u16 pop16(gb* g) { u16 v = r16(g, SP); SP+=2; return v; } // pop

// opcode handlers
// see: http://www.pastraiser.com/cpu/gameboy/gameboy_opcodes.html for reference
// _TODO: make pretty, i don't like this approach really, but it's easier to debug

// CB extension handler
// OK, this doesn't look nice, it's a hack to get a byte from u16 array with the right endianess
__device__ u8* ptrs(gb* g, u8 idx) {
  u8* _ptrs[8] = {&B, &C, &D, &E, &H, &L, 0, &A}; return _ptrs[idx];
}

#define PTR_REG(x) ptrs(g, (x))
// bitwise ops
// cb
__device__ u8 rlc(gb* g, u8 v) { // rotate left with carry
  u8 c = ((v >> 7) == 0x01); // carry if bit 7 set
  u8 r = (v << 1) | c; // shift and carry previous bit 7 into 0
  fH = 0; fN = 0; fZ = (r == 0); fC = c;
  return r;
}

__device__ u8 rrc(gb* g, u8 v) { // rotate right with carry
  u8 c = (v & 0x01); // carry if bit 0 set
  u8 r = (v >> 1) | (c << 7); // shift and carry previous bit 0 into 7
  fH = 0; fN = 0; fZ = (r == 0); fC = c;
  return r;
}

__device__ u8 rl(gb* g, u8 v) { // rotate left
  u8 c = ((v >> 7) == 0x01); // carry if bit 7 set
  u8 r = (0xff & (v << 1)) | fC;      // shift and carry from flags into 0
  if (r==0) {fZ = 1;} else {fZ = 0;}
  fH = 0; fN = 0; fC = c;
  return r;
}

__device__ u8 rr(gb* g, u8 v) { // rotate right
  u8 c = (v & 0x01); // carry if bit 0 set
  u8 r = (v >> 1) | (fC << 7); // shift and carry from flags into 7
  fH = 0; fN = 0; fZ = (r == 0); fC = c;
  return r;
}

__device__ u8 sla(gb* g, u8 v) { // shift left arithmetic
  u8 c = (v >> 7) & 0x1; // if bit 7 set
  u8 r = (v << 1);
  fH = 0; fN = 0; fZ = (r == 0); fC = c;
  return r;
}

__device__ u8 sra(gb* g, u8 v) { //shift right arithmetic
  u8 c = (v & 0x1); //if bit 0 set
  u8 r = (v >> 1) | (v & 0x80); // shift and extend sign
  fH = 0; fN = 0; fZ = (r == 0); fC = c;
  return r;
}

__device__ u8 srl(gb* g, u8 v) { //shift right logical
  u8 c = (v & 0x1); //if bit 0 set
  u8 r = (v >> 1); // shift
  fH = 0; fN = 0; fZ = (r == 0); fC = c;
  return r;
}

__device__ u8 swap(gb* g, u8 v) {
  fZ = (v==0); fC=0; fN=0; fH=0;
  return ( (v >> 4) | (v << 4) );
}

__device__ void bitchk(gb* g, u8 n, u8 v) {
  u8 r = ((v >> n) & 0x1) == 0;
  //fZ = 0 of bit was 1
  fN = 0; fH = 1; fZ = r;
}

// ops on accumulator (non-CB)
__device__ void rlca(gb* g) { A = rlc(g, A); fZ = 0;}
__device__ void rrca(gb* g) { A = rrc(g, A); fZ = 0;}
__device__ void  rla(gb* g) { A =  rl(g, A); fZ = 0;}
__device__ void  rra(gb* g) { A =  rr(g, A); fZ = 0;}


__device__ u8 inc8(gb* g, u8 v) {
  u8 r = v + 1;
  fZ = (r == 0); fH = ((v & 0x0f) + 1 > 0x0f); fN = 0;
  return r;
}

__device__ u8 dec8(gb* g, u8 v) {
  u8 r = v - 1;
  fZ = (r == 0); fH = ((v & 0x0f) == 0); fN = 1;
  return r;
}

// 8-bit alu ops
__device__ void _add8(gb* g, u8 v, u8 carry) {
  u8 c = carry ? fC : 0;
  u8 r = A;
  r = A + v + c;
  fH = (((A & 0xf) + (v & 0xf) + c) > 0xf) ? 1 : 0;
  fN = 0; fC = (((u16)(A) + (u16)(v) + (u16)(c)) > 0x00ff) ? 1 : 0;
  A = r;
  fZ = (A == 0);
}

__device__ void _sub8(gb * g, u8 v, u8 carry) {
  // use carry?
  u8 c = carry ? fC : 0;
  u8 r = A;
  r = A - v - c;
  // update flags
  fZ = (r == 0); fH = (((A & 0xf) < ((v & 0xf) + c))) ? 1 : 0; fN = 1;
  fC = (((u16)(A) < (u16)(v) + (u16)(c))) ? 1 : 0;
  A = r;
}

__device__ void add8(gb * g, u8 v) { _add8(g, v, 0); }
__device__ void adc8(gb * g, u8 v) { _add8(g, v, 1); }
__device__ void sub8(gb * g, u8 v) { _sub8(g, v, 0); }
__device__ void sbc8(gb * g, u8 v) { _sub8(g, v, 1); }
__device__ void and8(gb * g, u8 v) { A &= v; fZ = (A == 0); fH = 1; fC = 0; fN = 0; }
__device__ void or8 (gb * g, u8 v)  { A |= v; fZ = (A == 0); fH = 0; fC = 0; fN = 0; }
__device__ void xor8(gb * g, u8 v) { A ^= v; fZ = (A == 0); fH = 0; fC = 0; fN = 0; }
__device__ void cp8 (gb * g, u8 v)  { u8 r = A; _sub8(g, v, 0); A = r; }

// 8-bit alu
__device__ void alu(gb *g, u8 op) {
  u8 src_idx = op & 0x7; //last 3 bits are reg#
  u8 src = (((op >> 6) & 0x3) == 3) ? f8(g) : src_idx == 6 ? r8(g, HL) : (*PTR_REG(src_idx));
  u8 n = (op >> 3) & 0x07;

  switch (n) { // subgroup, bits xxNNNyyy
    case 0: add8(g, src); break; // 00000yyy
    case 1: adc8(g, src); break; // 00001yyy
    case 2: sub8(g, src); break; // 00010yyy
    case 3: sbc8(g, src); break; // 00011yyy
    case 4: and8(g, src); break; // 00100yyy
    case 5: xor8(g, src); break; // 00101yyy
    case 6: or8(g, src);  break; // 00110yyy
    case 7: cp8(g, src);  break; // 00111yyy
  };
}

// 8-bit inc/dec
__device__ void incdec(gb* g, u8 op) {
  u8 n = op & 0x3; //dec/inc
  u8 dst_idx = (op >> 3) & 0x7; //last 3 bits are reg#
  u8 src = dst_idx == 6 ? r8(g, HL) : (*PTR_REG(dst_idx));
  src = n ? dec8(g, src) : inc8(g, src);
  if (dst_idx != 6) *PTR_REG(dst_idx) = src; else w8(g, HL, src);
}

// register - register load
__device__ void ldrr(gb* g, u8 op) {
  u8 src_idx = op & 0x7; //last 3 bits are reg#
  u8 dst_idx = (op >> 3) & 0x7; //last 3 bits are reg#
  u8 src = (((op >> 6) & 0x3) == 0) ? f8(g) : src_idx == 6 ? r8(g, HL) : (*PTR_REG(src_idx));
  if (dst_idx != 6) *PTR_REG(dst_idx) = src; else w8(g, HL, src);
}

__device__ void cb_ex(gb* g, u8 x) {

#ifdef DEBUG
  //printf("CB_ex %02x\n", x);
#endif

  u8 src_idx = x & 0x7; //last 3 bits are reg#
  u8 src = src_idx == 6 ? r8(g, HL) : (*PTR_REG(src_idx));
  u8 op_group = (x >> 6) & 0x03;
  u8 n = (x >> 3) & 0x07;
  u8 res = src;

  switch (op_group) {
    case 0:  // opcode == 00xxxyyy
      switch (n) { // subgroup, bits xxNNNyyy
        case 0: res = rlc (g, src);  break; // 00000yyy
        case 1: res = rrc (g, src);  break; // 00001yyy
        case 2: res = rl  (g, src);   break; // 00010yyy
        case 3: res = rr  (g, src);   break; // 00011yyy
        case 4: res = sla (g, src);  break; // 00100yyy
        case 5: res = sra (g, src);  break; // 00101yyy
        case 6: res = swap(g, src); break; // 00110yyy
        case 7: res = srl (g, src);  break; // 00111yyy
      }; break;
    case 1:  // opcode == 01xxxyyy, test bit n
      bitchk(g, n, src); break;
    case 2:  // opcode == 10xxxyyy, clear bit n
      res &= ~(1<<n); break;
    case 3:  // opcode == 11xxxyyy, set bit n
      res |= (1<<n);  break;
  }

  if (src_idx != 6) *PTR_REG(src_idx) = res; else w8(g, HL, res);
  u8 mcycl = (src_idx == 6)  ? 16 : 8; // 16 cycles if hl
  g->cpu_ticks += mcycl;
}

// JUMPS
__device__ void jr  (gb *g) { PC += (s8)(f8(g)); } // jump relative
__device__ void jp  (gb *g) { PC = r16(g, PC); } // jump absolute
__device__ void jphl(gb *g) { PC = HL; }
__device__ void call(gb *g) { push16(g, PC+2); PC=r16(g,PC); } // uncoditional call
__device__ void ret (gb *g) { PC = pop16(g); } // return from call
__device__ void rst (gb *g, u8 v) { push16(g, PC); PC = (u16)(v); } // reset

// 00 - 0f
__device__ void x00(gb *g, u8 op) { /* nop */ }
__device__ void x01(gb *g, u8 op) { BC = r16(g, PC); PC+=2; }
__device__ void x02(gb *g, u8 op) { w8(g,BC,A); }
__device__ void x03(gb *g, u8 op) { BC++; }
__device__ void x04(gb *g, u8 op) { incdec(g, op); }
__device__ void x05(gb *g, u8 op) { incdec(g, op); }
__device__ void x06(gb *g, u8 op) { ldrr(g, op); }
__device__ void x07(gb *g, u8 op) { rlca(g); }
__device__ void x08(gb *g, u8 op) { u16 a = r16(g, PC); w16(g, a, SP); PC+=2; } // LD (a16), SP
__device__ void x09(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x0a(gb *g, u8 op) { A = r8(g,BC); }
__device__ void x0b(gb *g, u8 op) { BC--; }
__device__ void x0c(gb *g, u8 op) { incdec(g, op); }
__device__ void x0d(gb *g, u8 op) { incdec(g, op); }
__device__ void x0e(gb *g, u8 op) { ldrr(g, op);                       } // ld c, imm8
__device__ void x0f(gb *g, u8 op) { rrca(g); }
// 10 - 1f
__device__ void x10(gb *g, u8 op) { g->stopped = 1; }
__device__ void x11(gb *g, u8 op) { DE = r16(g, PC); PC+=2; }
__device__ void x12(gb *g, u8 op) { w8(g,DE,A); }
__device__ void x13(gb *g, u8 op) { DE++; }
__device__ void x14(gb *g, u8 op) { incdec(g, op); }
__device__ void x15(gb *g, u8 op) { incdec(g, op); }
__device__ void x16(gb *g, u8 op) { ldrr(g, op); }
__device__ void x17(gb *g, u8 op) { rla(g); }
__device__ void x18(gb *g, u8 op) { jr(g); } // jr, s8
__device__ void x19(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x1a(gb *g, u8 op) { A = r8(g, DE); }
__device__ void x1b(gb *g, u8 op) { DE--; }
__device__ void x1c(gb *g, u8 op) { incdec(g, op); }
__device__ void x1d(gb *g, u8 op) { incdec(g, op); }
__device__ void x1e(gb *g, u8 op) { ldrr(g, op);                       } // ld e, imm8
__device__ void x1f(gb *g, u8 op) { rra(g); }
// 20 - 2f
__device__ void x20(gb *g, u8 op) { if (!fZ) jr(g); else { PC+=1; }  } // jr nz, s8
__device__ void x21(gb *g, u8 op) { HL = r16(g, PC); PC+=2;          } // ld hl, imm16
__device__ void x22(gb *g, u8 op) { w8(g, HL++, A); }
__device__ void x23(gb *g, u8 op) { HL++; }
__device__ void x24(gb *g, u8 op) { incdec(g, op); }
__device__ void x25(gb *g, u8 op) { incdec(g, op); }
__device__ void x26(gb *g, u8 op) { ldrr(g, op); }
__device__ void x27(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x28(gb *g, u8 op) { if (fZ)  jr(g); else { PC+=1; }   } // jr z, s8
__device__ void x29(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x2a(gb *g, u8 op) { A = r8(g, HL++); }
__device__ void x2b(gb *g, u8 op) { HL--; }
__device__ void x2c(gb *g, u8 op) { incdec(g, op); }
__device__ void x2d(gb *g, u8 op) { incdec(g, op); }
__device__ void x2e(gb *g, u8 op) { ldrr(g, op);                       } // ld l, imm8
__device__ void x2f(gb *g, u8 op) { g->unimpl = 1; }
// 30 - 3f
__device__ void x30(gb *g, u8 op) { if (!fC) jr(g); else { PC+=1; }  } // jr nc,s8
__device__ void x31(gb *g, u8 op) { SP = r16(g, PC); PC+=2;          } // ld sp, imm16
__device__ void x32(gb *g, u8 op) { w8(g, HL--, A); }
__device__ void x33(gb *g, u8 op) { SP++; }
__device__ void x34(gb *g, u8 op) { incdec(g, op); }
__device__ void x35(gb *g, u8 op) { incdec(g, op); }
__device__ void x36(gb *g, u8 op) { ldrr(g, op); }
__device__ void x37(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x38(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x39(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x3a(gb *g, u8 op) { g->unimpl = 1; }
__device__ void x3b(gb *g, u8 op) { SP--; }
__device__ void x3c(gb *g, u8 op) { incdec(g, op); }
__device__ void x3d(gb *g, u8 op) { incdec(g, op); }
__device__ void x3e(gb *g, u8 op) { ldrr(g, op);                       } // ld a, imm8
__device__ void x3f(gb *g, u8 op) { g->unimpl = 1; }

// these are all reg-reg loads
// 40 - 4f
__device__ void x40(gb *g, u8 op) { ldrr(g, op); }; __device__ void x41(gb *g, u8 op) { ldrr(g, op); };
__device__ void x42(gb *g, u8 op) { ldrr(g, op); }; __device__ void x43(gb *g, u8 op) { ldrr(g, op); };
__device__ void x44(gb *g, u8 op) { ldrr(g, op); }; __device__ void x45(gb *g, u8 op) { ldrr(g, op); };
__device__ void x46(gb *g, u8 op) { ldrr(g, op); }; __device__ void x47(gb *g, u8 op) { ldrr(g, op); };
__device__ void x48(gb *g, u8 op) { ldrr(g, op); }; __device__ void x49(gb *g, u8 op) { ldrr(g, op); };
__device__ void x4a(gb *g, u8 op) { ldrr(g, op); }; __device__ void x4b(gb *g, u8 op) { ldrr(g, op); };
__device__ void x4c(gb *g, u8 op) { ldrr(g, op); }; __device__ void x4d(gb *g, u8 op) { ldrr(g, op); };
__device__ void x4e(gb *g, u8 op) { ldrr(g, op); }; __device__ void x4f(gb *g, u8 op) { ldrr(g, op); };
// 50 - 5f
__device__ void x50(gb *g, u8 op) { ldrr(g, op); }; __device__ void x51(gb *g, u8 op) { ldrr(g, op); };
__device__ void x52(gb *g, u8 op) { ldrr(g, op); }; __device__ void x53(gb *g, u8 op) { ldrr(g, op); };
__device__ void x54(gb *g, u8 op) { ldrr(g, op); }; __device__ void x55(gb *g, u8 op) { ldrr(g, op); };
__device__ void x56(gb *g, u8 op) { ldrr(g, op); }; __device__ void x57(gb *g, u8 op) { ldrr(g, op); };
__device__ void x58(gb *g, u8 op) { ldrr(g, op); }; __device__ void x59(gb *g, u8 op) { ldrr(g, op); };
__device__ void x5a(gb *g, u8 op) { ldrr(g, op); }; __device__ void x5b(gb *g, u8 op) { ldrr(g, op); };
__device__ void x5c(gb *g, u8 op) { ldrr(g, op); }; __device__ void x5d(gb *g, u8 op) { ldrr(g, op); };
__device__ void x5e(gb *g, u8 op) { ldrr(g, op); }; __device__ void x5f(gb *g, u8 op) { ldrr(g, op); };
// 60 - 6f
__device__ void x60(gb *g, u8 op) { ldrr(g, op); }; __device__ void x61(gb *g, u8 op) { ldrr(g, op); };
__device__ void x62(gb *g, u8 op) { ldrr(g, op); }; __device__ void x63(gb *g, u8 op) { ldrr(g, op); };
__device__ void x64(gb *g, u8 op) { ldrr(g, op); }; __device__ void x65(gb *g, u8 op) { ldrr(g, op); };
__device__ void x66(gb *g, u8 op) { ldrr(g, op); }; __device__ void x67(gb *g, u8 op) { ldrr(g, op); };
__device__ void x68(gb *g, u8 op) { ldrr(g, op); }; __device__ void x69(gb *g, u8 op) { ldrr(g, op); };
__device__ void x6a(gb *g, u8 op) { ldrr(g, op); }; __device__ void x6b(gb *g, u8 op) { ldrr(g, op); };
__device__ void x6c(gb *g, u8 op) { ldrr(g, op); }; __device__ void x6d(gb *g, u8 op) { ldrr(g, op); };
__device__ void x6e(gb *g, u8 op) { ldrr(g, op); }; __device__ void x6f(gb *g, u8 op) { ldrr(g, op); };
// 70 - 7f
__device__ void x70(gb *g, u8 op) { ldrr(g, op); }; __device__ void x71(gb *g, u8 op) { ldrr(g, op); };
__device__ void x72(gb *g, u8 op) { ldrr(g, op); }; __device__ void x73(gb *g, u8 op) { ldrr(g, op); };
__device__ void x74(gb *g, u8 op) { ldrr(g, op); }; __device__ void x75(gb *g, u8 op) { ldrr(g, op); };
__device__ void x76(gb *g, u8 op) { /* HALT */   }; __device__ void x77(gb *g, u8 op) { ldrr(g, op); };
__device__ void x78(gb *g, u8 op) { ldrr(g, op); }; __device__ void x79(gb *g, u8 op) { ldrr(g, op); };
__device__ void x7a(gb *g, u8 op) { ldrr(g, op); }; __device__ void x7b(gb *g, u8 op) { ldrr(g, op); };
__device__ void x7c(gb *g, u8 op) { ldrr(g, op); }; __device__ void x7d(gb *g, u8 op) { ldrr(g, op); };
__device__ void x7e(gb *g, u8 op) { ldrr(g, op); }; __device__ void x7f(gb *g, u8 op) { ldrr(g, op); };

// reg-reg alu
// 80 - 8f
__device__ void x80(gb *g, u8 op) { alu(g, op);  }; __device__ void x81(gb *g, u8 op) { alu(g, op);  };
__device__ void x82(gb *g, u8 op) { alu(g, op);  }; __device__ void x83(gb *g, u8 op) { alu(g, op);  };
__device__ void x84(gb *g, u8 op) { alu(g, op);  }; __device__ void x85(gb *g, u8 op) { alu(g, op);  };
__device__ void x86(gb *g, u8 op) { alu(g, op);  }; __device__ void x87(gb *g, u8 op) { alu(g, op);  };
__device__ void x88(gb *g, u8 op) { alu(g, op);  }; __device__ void x89(gb *g, u8 op) { alu(g, op);  };
__device__ void x8a(gb *g, u8 op) { alu(g, op);  }; __device__ void x8b(gb *g, u8 op) { alu(g, op);  };
__device__ void x8c(gb *g, u8 op) { alu(g, op);  }; __device__ void x8d(gb *g, u8 op) { alu(g, op);  };
__device__ void x8e(gb *g, u8 op) { alu(g, op);  }; __device__ void x8f(gb *g, u8 op) { alu(g, op);  };
// 90 - 9f;
__device__ void x90(gb *g, u8 op) { alu(g, op);  }; __device__ void x91(gb *g, u8 op) { alu(g, op);  };
__device__ void x92(gb *g, u8 op) { alu(g, op);  }; __device__ void x93(gb *g, u8 op) { alu(g, op);  };
__device__ void x94(gb *g, u8 op) { alu(g, op);  }; __device__ void x95(gb *g, u8 op) { alu(g, op);  };
__device__ void x96(gb *g, u8 op) { alu(g, op);  }; __device__ void x97(gb *g, u8 op) { alu(g, op);  };
__device__ void x98(gb *g, u8 op) { alu(g, op);  }; __device__ void x99(gb *g, u8 op) { alu(g, op);  };
__device__ void x9a(gb *g, u8 op) { alu(g, op);  }; __device__ void x9b(gb *g, u8 op) { alu(g, op);  };
__device__ void x9c(gb *g, u8 op) { alu(g, op);  }; __device__ void x9d(gb *g, u8 op) { alu(g, op);  };
__device__ void x9e(gb *g, u8 op) { alu(g, op);  }; __device__ void x9f(gb *g, u8 op) { alu(g, op);  };
// a0 - af;
__device__ void xa0(gb *g, u8 op) { alu(g, op);  }; __device__ void xa1(gb *g, u8 op) { alu(g, op);  };
__device__ void xa2(gb *g, u8 op) { alu(g, op);  }; __device__ void xa3(gb *g, u8 op) { alu(g, op);  };
__device__ void xa4(gb *g, u8 op) { alu(g, op);  }; __device__ void xa5(gb *g, u8 op) { alu(g, op);  };
__device__ void xa6(gb *g, u8 op) { alu(g, op);  }; __device__ void xa7(gb *g, u8 op) { alu(g, op);  };
__device__ void xa8(gb *g, u8 op) { alu(g, op);  }; __device__ void xa9(gb *g, u8 op) { alu(g, op);  };
__device__ void xaa(gb *g, u8 op) { alu(g, op);  }; __device__ void xab(gb *g, u8 op) { alu(g, op);  };
__device__ void xac(gb *g, u8 op) { alu(g, op);  }; __device__ void xad(gb *g, u8 op) { alu(g, op);  };
__device__ void xae(gb *g, u8 op) { alu(g, op);  }; __device__ void xaf(gb *g, u8 op) { alu(g, op);  };
// b0 - bf;
__device__ void xb0(gb *g, u8 op) { alu(g, op);  }; __device__ void xb1(gb *g, u8 op) { alu(g, op);  };
__device__ void xb2(gb *g, u8 op) { alu(g, op);  }; __device__ void xb3(gb *g, u8 op) { alu(g, op);  };
__device__ void xb4(gb *g, u8 op) { alu(g, op);  }; __device__ void xb5(gb *g, u8 op) { alu(g, op);  };
__device__ void xb6(gb *g, u8 op) { alu(g, op);  }; __device__ void xb7(gb *g, u8 op) { alu(g, op);  };
__device__ void xb8(gb *g, u8 op) { alu(g, op);  }; __device__ void xb9(gb *g, u8 op) { alu(g, op);  };
__device__ void xba(gb *g, u8 op) { alu(g, op);  }; __device__ void xbb(gb *g, u8 op) { alu(g, op);  };
__device__ void xbc(gb *g, u8 op) { alu(g, op);  }; __device__ void xbd(gb *g, u8 op) { alu(g, op);  };
__device__ void xbe(gb *g, u8 op) { alu(g, op);  }; __device__ void xbf(gb *g, u8 op) { alu(g, op);  };
// c0 - cf
__device__ void xc0(gb *g, u8 op) { if (!fZ) ret(g); else g-> cpu_ticks -= 12; } //ret nz
__device__ void xc1(gb *g, u8 op) { BC = pop16(g); }
__device__ void xc2(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xc3(gb *g, u8 op) { jp(g); }
__device__ void xc4(gb *g, u8 op) { if (!fZ) call(g); else { g->cpu_ticks -= 12; PC+=2; } } // call nz, imm16
__device__ void xc5(gb *g, u8 op) { push16(g, BC); }
__device__ void xc6(gb *g, u8 op) { alu(g, op); }
__device__ void xc7(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xc8(gb *g, u8 op) { if (fZ) ret(g); else g->cpu_ticks -= 12; } // ret z
__device__ void xc9(gb *g, u8 op) { ret(g); }
__device__ void xca(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xcb(gb *g, u8 op) { cb_ex(g, f8(g)); } // CB extension
__device__ void xcc(gb *g, u8 op) { if (fZ) call(g); else { g->cpu_ticks -= 12; PC+=2; } } // call z imm16
__device__ void xcd(gb *g, u8 op) { call(g); }
__device__ void xce(gb *g, u8 op) { alu(g, op); }
__device__ void xcf(gb *g, u8 op) { g->unimpl = 1; }
// d0 - df
__device__ void xd0(gb *g, u8 op) { if (!fC) ret(g); else g->cpu_ticks -= 12; } // ret nc
__device__ void xd1(gb *g, u8 op) { DE = pop16(g); }
__device__ void xd2(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xd3(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xd4(gb *g, u8 op) { if (!fC) call(g); else { g->cpu_ticks -= 12; PC+=2; } } // call nc imm 16
__device__ void xd5(gb *g, u8 op) { push16(g,DE); }
__device__ void xd6(gb *g, u8 op) { alu(g, op); }
__device__ void xd7(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xd8(gb *g, u8 op) { if (fC) ret(g); else g-> cpu_ticks -= 12; } // ret c
__device__ void xd9(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xda(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xdb(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xdc(gb *g, u8 op) { if (fC) call(g); else { g->cpu_ticks -= 12; PC+=2; } } // call c imm 16
__device__ void xdd(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xde(gb *g, u8 op) { alu(g, op); }
__device__ void xdf(gb *g, u8 op) { g->unimpl = 1; }
// e0 - ef
__device__ void xe0(gb *g, u8 op) { w8(g, 0xff00 | f8(g), A); }
__device__ void xe1(gb *g, u8 op) { HL = pop16(g); }
__device__ void xe2(gb *g, u8 op) { w8(g, 0xff00 | C, A); }
__device__ void xe3(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xe4(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xe5(gb *g, u8 op) { push16(g,HL); }
__device__ void xe6(gb *g, u8 op) { alu(g, op); }
__device__ void xe7(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xe8(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xe9(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xea(gb *g, u8 op) { w8(g, f16(g), A); }
__device__ void xeb(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xec(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xed(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xee(gb *g, u8 op) { alu(g, op); }
__device__ void xef(gb *g, u8 op) { g->unimpl = 1; }
// f0 - ff
__device__ void xf0(gb *g, u8 op) { A = r8(g, 0xff00 | f8(g));    }
__device__ void xf1(gb *g, u8 op) { AF = pop16(g) & 0xfff0; }
__device__ void xf2(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xf3(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xf4(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xf5(gb *g, u8 op) { push16(g,AF); }
__device__ void xf6(gb *g, u8 op) { alu(g, op);; }
__device__ void xf7(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xf8(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xf9(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xfa(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xfb(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xfc(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xfd(gb *g, u8 op) { g->unimpl = 1; }
__device__ void xfe(gb *g, u8 op) { alu(g, op); }
__device__ void xff(gb *g, u8 op) { g->unimpl = 1; }

// assign ptrs to opcode handlers
// _TODO: make pretty
__device__ void* ops[256] =
{ &x00, &x01, &x02, &x03, &x04, &x05, &x06, &x07, &x08, &x09, &x0a, &x0b, &x0c, &x0d, &x0e, &x0f,
  &x10, &x11, &x12, &x13, &x14, &x15, &x16, &x17, &x18, &x19, &x1a, &x1b, &x1c, &x1d, &x1e, &x1f,
  &x20, &x21, &x22, &x23, &x24, &x25, &x26, &x27, &x28, &x29, &x2a, &x2b, &x2c, &x2d, &x2e, &x2f,
  &x30, &x31, &x32, &x33, &x34, &x35, &x36, &x37, &x38, &x39, &x3a, &x3b, &x3c, &x3d, &x3e, &x3f,
  &x40, &x41, &x42, &x43, &x44, &x45, &x46, &x47, &x48, &x49, &x4a, &x4b, &x4c, &x4d, &x4e, &x4f,
  &x50, &x51, &x52, &x53, &x54, &x55, &x56, &x57, &x58, &x59, &x5a, &x5b, &x5c, &x5d, &x5e, &x5f,
  &x60, &x61, &x62, &x63, &x64, &x65, &x66, &x67, &x68, &x69, &x6a, &x6b, &x6c, &x6d, &x6e, &x6f,
  &x70, &x71, &x72, &x73, &x74, &x75, &x76, &x77, &x78, &x79, &x7a, &x7b, &x7c, &x7d, &x7e, &x7f,
  &x80, &x81, &x82, &x83, &x84, &x85, &x86, &x87, &x88, &x89, &x8a, &x8b, &x8c, &x8d, &x8e, &x8f,
  &x90, &x91, &x92, &x93, &x94, &x95, &x96, &x97, &x98, &x99, &x9a, &x9b, &x9c, &x9d, &x9e, &x9f,
  &xa0, &xa1, &xa2, &xa3, &xa4, &xa5, &xa6, &xa7, &xa8, &xa9, &xaa, &xab, &xac, &xad, &xae, &xaf,
  &xb0, &xb1, &xb2, &xb3, &xb4, &xb5, &xb6, &xb7, &xb8, &xb9, &xba, &xbb, &xbc, &xbd, &xbe, &xbf,
  &xc0, &xc1, &xc2, &xc3, &xc4, &xc5, &xc6, &xc7, &xc8, &xc9, &xca, &xcb, &xcc, &xcd, &xce, &xcf,
  &xd0, &xd1, &xd2, &xd3, &xd4, &xd5, &xd6, &xd7, &xd8, &xd9, &xda, &xdb, &xdc, &xdd, &xde, &xdf,
  &xe0, &xe1, &xe2, &xe3, &xe4, &xe5, &xe6, &xe7, &xe8, &xe9, &xea, &xeb, &xec, &xed, &xee, &xef,
  &xf0, &xf1, &xf2, &xf3, &xf4, &xf5, &xf6, &xf7, &xf8, &xf9, &xfa, &xfb, &xfc, &xfd, &xfe, &xff };

__device__ void ppu_draw_bg(gb *g) {

  u8 gpu_win_on = (REG_LCDC >> 5) & 0x1;//off,on
  u8 lcd_on     = (REG_LCDC >> 7) & 0x1;//off,on

  if (lcd_on) {
    u8 bgy = (REG_SCANLINE+REG_SCY);
    u16 bgtiley = (((u16)bgy) >> 3) & 31;
    s32 winy = gpu_win_on ? (REG_SCANLINE) - (REG_WINY) : -1;
    u16 wintiley = (((u16)winy) >> 3) & 31;
    for (u8 x=0; x<160; x++) {
      u32 bgx = (u32)REG_SCX + (u32)x;
      s32 winx = -(((s32)REG_WINX) - 7) + x;
      u16 tilemapbase;
      u16 tilex, tiley, pixelx, pixely;
      u8 gpu_bgmap =   (REG_LCDC >> 3) & 0x1;//9800-9bff, 9c00-9fff
      u8 gpu_tilemap = (REG_LCDC >> 4) & 0x1;//8800-97ff, 8000-8fff
      u8 gpu_drawbg = (REG_LCDC >> 0) & 0x1;//off,on
      u16 tilebase = (gpu_tilemap ? 0x8000 : 0x8800);
      if (winx >= 0 && winy >=0 ) { // draw window
        u8 gpu_win_map=(REG_LCDC >> 6) & 0x1;//9800-9bff, 9c00-9fff
        tilemapbase=gpu_win_map ? 0x9c00 : 0x9800;
        tiley=wintiley; tilex=(((u16)winx) >> 3); pixely=((u16)winy) & 0x7; pixelx=((u8)winx) & 0x7;
      } else if (gpu_drawbg) { // draw bg
        tilemapbase=gpu_bgmap ? 0x9c00 : 0x9800;
        tiley=bgtiley; tilex=(((u16)bgx) >> 3) & 31; pixely=((u16)bgy) & 0x7; pixelx=((u8)bgx) & 0x7;
      } else {};

      u8 _tilenr = r8(g,tilemapbase + tiley * 32 + tilex);
      u16 tilenr, tileaddress;
      if (tilebase == 0x8800) {
        int8_t nr_s = (s8)_tilenr; s16 nr_s16 = (s16)nr_s + 128; tilenr = (u16)nr_s16;
      }
      else { tilenr = (u16)_tilenr; }
      tileaddress = tilenr * 16 + tilebase;

      u16 a0 = tileaddress + pixely*2;
      u8 data0 = r8(g,a0);
      u8 data1 = r8(g,a0+1);
      u8 color0_idx = ((data0 >> (7-pixelx)) & 0x1);
      u8 color1_idx = ((data1 >> (7-pixelx)) & 0x1);

      u8 color_idx = color0_idx + color1_idx*2;
      //u8 r,g,b;
      u8 color = (REG_BGRDPAL>>(color_idx*2))&0x3;
      //if (color > 0) printf("pix <= %d, %04x x = %d y = %d\n", color, a0, x, REG_SCANLINE);
      //bgprio[x] = color_idx;
      //if (color == 0) {r=255; g=255; b=255;}
      //if (color == 1) {r=192; g=192; b=192;}
      //if (color == 2) {r=96; g=96; b=96;}
      //if (color == 3) {r=0; g=0; b=0;}

      u32 screen_off = ((u32)REG_SCANLINE)*160 + x;
      if (screen_off < 144*160) {
        g->pix[screen_off] = color;
      }
    }
  } else { } // lcd off

}

__device__ void ppu_draw_sprites(gb *g) { }
__device__ void ppu_renderscan(gb *g) { ppu_draw_bg(g); ppu_draw_sprites(g); }
__device__ void blit(gb *g) { g->frame_no++; }
__device__ void check_interrupt_lyc(gb *g) { }

__device__ void ppu_change_mode(gb* g, u8 new_mode) {

  g->ppu_mode = new_mode;

  //REG_LCDSTAT &= ~(0x3); REG_LCDSTAT |= g->ppu_mode & 0x3;
  //REG_LCDSTAT &= ~(0x4); REG_LCDSTAT |= ((REG_SCANLINE == REG_LYC) << 2);

  //u8 irq = 1;
  //u8 m0e = (REG_LCDSTAT >> 3) & 0x1; //hblank int
  //u8 m1e = (REG_LCDSTAT >> 4) & 0x1; //vblank int
  //u8 m2e = (REG_LCDSTAT >> 5) & 0x1; //oam int
  switch (g->ppu_mode) {
    case 0: //irq &= m0e;
            //if (g->enable_ppu)
            ppu_renderscan(g);
            //gpu_hblanking = 1;
            break;
    case 1: /*irq &= m1e; REG_INTF |= 0x01;*/ break;
    case 2: /*irq &= m2e; oam_ram();*/ break;
    case 3: /*irq = 0;*/ break;
  }

  //if (irq) REG_INTF |= 0x2;
}

__device__ void ppu_step(gb* g, u32 ticks) {

  g->ppu_mode_clk += ticks;

  //printf("scanline %d ppu clks %d mode %d\n", REG_SCANLINE, g->ppu_mode_clk, g->ppu_mode);
  if (g->ppu_mode_clk >= 456) { g->ppu_mode_clk -= 456; REG_SCANLINE = (REG_SCANLINE + 1) % 154; }

  check_interrupt_lyc(g);

  if ((REG_SCANLINE >= 144) && (g->ppu_mode != 1)) { ppu_change_mode(g, 1); blit(g); } // vblank
  else {
    // not vblank
    if (g->ppu_mode_clk <= 80) {
      if (g->ppu_mode != 2) ppu_change_mode(g,2); // oam scan
    } else if (g->ppu_mode_clk <= 252) {
      if (g->ppu_mode != 3) ppu_change_mode(g,3); // render line
    } else {
      if (g->ppu_mode != 0) ppu_change_mode(g,0); // hblank
    }
  }

}
__global__ void exec(gb* _g_regs, u8 *prog, u16* _g_rand, int prog_len, int steps, int num_threads) {
  int i = blockDim.x * blockIdx.x + threadIdx.x; // thread idx

  u8 op = 0;

  if (i < num_threads) {

    gb *g = &_g_regs[i]; // local copy of regs
    g->rom = prog;

    for (int j = 0; j < steps; ++j) {

      op = prog[PC]; PC = (PC + 1) % prog_len;
      ((void(*)(gb*,u8))ops[op])(g,op);
      g->cpu_instr++; u8 cpu_cycles = mcycles[op];
      g->cpu_ticks += cpu_cycles;
      ppu_step(g, cpu_cycles);
        printf("I %08d M %08d OP = %02x BC = %04x DE = %04x HL = %04x AF = %04x SP = %04x PC = %04x\n", g->cpu_instr, g->cpu_ticks, op, BC, DE, HL, AF, SP, PC);
      if (g->unimpl) {
        //printf("!!!!    %05d [%04d, %05d] j %5d, ins %08d tic %08d, op %02x BC = %04x DE = %04x HL = %04x AF = %04x SP = %04x PC = %04x ff25 %02x\n",i, blockIdx.x, threadIdx.x, j, g->cpu_instr, g->cpu_ticks, op, BC, DE, HL, AF, SP, PC, g->hram[0x11]); return;
      }
      if (j == steps-1) {
        //printf("last    %05d [%04d, %05d] j %5d, ins %08d tic %08d, op %02x BC = %04x DE = %04x HL = %04x AF = %04x SP = %04x PC = %04x ff25 %02x\n",i, blockIdx.x, threadIdx.x, j, g->cpu_instr, g->cpu_ticks, op, BC, DE, HL, AF, SP, PC, g->hram[0x11]);
      }
      //if (PC == 0x00fe) {
      //  // print mid screen
      //  for (int row=62; row < 82; row++) { printf("%3d: ", row); for (int i = 0; i < 160; i++) { printf("%c", g->pix[160*row + i] > 0 ? '#' : '.'); if (i==159) printf("\n"); } } }
      //if (g->unimpl == 1 || PC >= 0x00fe) { return; };
    }
  }
}

int main(int argc, char **argv) {

  hipError_t err = hipSuccess; // for checking CUDA errors

  // Print the vector length to be used, and compute its size
  int num_blocks = 1; int threads_per_block = 1; int prog_len = 32768; int iters = 4; int steps = 1024;

  // override defaults
  if (argc >= 2) num_blocks = atoi(argv[1]);
  if (argc >= 3) threads_per_block = atoi(argv[2]);
  if (argc >= 4) iters = atoi(argv[3]);
  if (argc >= 5) steps = atoi(argv[4]);

  int num_threads = num_blocks * threads_per_block;

  // allocate mem for prog
  u8    *h_prog  = (u8 *)   malloc(prog_len * sizeof(u8));
  const char* fname = "./06-ldrr.gb";
  if (read_cart(h_prog, fname) < 0) { printf("could read %s\n", fname); free(h_prog); return; }

  printf("  main: running %d blocks * %d threads (%d threads total)\n", num_blocks, threads_per_block, num_threads);

  // allocate gb registers / state
  gb    *h_in_regs   = (gb *) malloc(num_threads * sizeof(gb));
  gb    *h_out_regs  = (gb *) malloc(num_threads * sizeof(gb));
  u16   *h_rand      = (u16*) malloc(num_threads * sizeof(u16));

  if (h_rand == NULL || h_in_regs == NULL || h_out_regs == NULL || h_prog == NULL) { fprintf(stderr, "Failed to allocate host mem!\n"); exit(-1); }

  srand(get_time());
  for (int i = 0; i < num_threads; ++i) { h_rand[i] = rand(); }

  // alloc gpu mem
  printf("  main: allocating %zu device bytes\n", prog_len * sizeof(u8) + num_threads * sizeof(gb));

  u8 *d_prog = NULL; gb* d_regs = NULL; u16* d_rand = NULL;
  err = hipMalloc((void **)&d_prog, prog_len    * sizeof(u8) ); CHECK_ERR_CUDA(err);
  err = hipMalloc((void **)&d_regs, num_threads * sizeof(gb) ); CHECK_ERR_CUDA(err);
  err = hipMalloc((void **)&d_rand, num_threads * sizeof(u16)); CHECK_ERR_CUDA(err);

  printf("  main: copying host -> device\n");
  err = hipMemcpy(d_prog, h_prog,    sizeof(u8 ) * prog_len,    hipMemcpyHostToDevice);  CHECK_ERR_CUDA(err);
  err = hipMemcpy(d_regs, h_in_regs, sizeof(gb ) * num_threads, hipMemcpyHostToDevice);  CHECK_ERR_CUDA(err);
  err = hipMemcpy(d_rand, h_rand,    sizeof(u16) * num_threads, hipMemcpyHostToDevice);  CHECK_ERR_CUDA(err);

  for (int j = 0; j < iters; j++ ) {

    //printf("  main: %4d running kernel\n", j);
    hipDeviceSynchronize();
    double start_time = get_time();
    exec<<<num_blocks, threads_per_block>>>(d_regs, d_prog, d_rand, prog_len, steps, num_threads);
    hipDeviceSynchronize();

    double walltime = get_time() - start_time;
    err = hipGetLastError(); CHECK_ERR_CUDA(err);
    //printf("  main: kernel time = %.6f s, %2.6f us/step, %5.3f MHz\n", walltime, 1e6 * (walltime/(steps * num_threads)), ((steps * num_threads)/walltime)/1e6);
  }

  printf("  main: copying device -> host\n");
  err = hipMemcpy(h_out_regs, d_regs, sizeof(gb) * num_threads, hipMemcpyDeviceToHost); CHECK_ERR_CUDA(err);
  printf("  main: freeing memory\n");

  // free gpu mem
  err = hipFree(d_prog); CHECK_ERR_CUDA(err);
  err = hipFree(d_regs); CHECK_ERR_CUDA(err);
  err = hipFree(d_rand); CHECK_ERR_CUDA(err);

  // free host mem
  free(h_in_regs); free(h_out_regs); free(h_prog); free(h_rand);

  printf("  main: done.\n");

  return 0;
}
